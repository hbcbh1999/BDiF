#include "hip/hip_runtime.h"
/*
Copyright: Copyright (C) 2015 Baruch College - All Rights Reserved
Description: CVA Main file
Author: Weiyi Chen, weiyi.chen@baruchmail.cuny.edu
*/

// thrust includes

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>
#include <thrust/reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/execution_policy.h>

// boost includes

#include <boost/serialization/vector.hpp>
#include <boost/archive/binary_oarchive.hpp>
#include <boost/archive/binary_iarchive.hpp>

// cuda includes

#include <nvml.h>
#include <omp.h>

// C++ includes

#include <unistd.h>
#include <fstream>

// local includes

#include "counterparty.h"
#include "gpu_helper.h"
#include "logFile.h"
#include "output.h"
#include "params.h"
#include "swap.h"
#include "timer.h"

int main(int argc, char **argv) {

    // read the simulation & model parameters and store them in data structures
    
    std::ifstream params_file(argv[1]);
    Params in = Params(params_file); 
    params_file.close();

    // open log file and write initial output

    Output::file() = fopen(in.log_file.c_str(), "a");
    LogFile::ReportLevel() = stringToLevel(in.log_level);
    LOG(OFF) << "EXECUTE\t" << argv[0];
    LOG(INFO) << "CONFIG\t" << argv[1];

    // Divide to CPU and GPU, write openmp info

#ifdef _CPU_
    LOG(OFF) << "CUDA SYSTEM\tOpenMP (CPU)";
    LOG(INFO) << "NUM_PROCESSORS\t" << omp_get_num_procs();
    LOG(INFO) << "NUM_THREADS\t" << omp_get_max_threads();
#endif

#ifdef _GPU_
    LOG(OFF) << "CUDA SYSTEM\tCUDA (GPU)";
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    LOG(INFO) << "NUM_GPUs\t" << num_gpus;
    for (unsigned i = 0; i < num_gpus; ++i) {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        LOG(INFO) << "   GPU " << i << " : " << dprop.name;
    }
    nvmlInit();
    char version[NVML_SYSTEM_NVML_VERSION_BUFFER_SIZE];
    if (nvmlSystemGetDriverVersion(version, NVML_SYSTEM_NVML_VERSION_BUFFER_SIZE) == NVML_SUCCESS) {
        LOG(INFO) << "chenggong : " << version;
    }
#endif

// TODO: how to get CUDA version, Thrust version, GPU memory usage info
// one possible way may be using system()
// cat /proc/cpuinfo | grep "model name" | awk -F: '{print $2}'
//=========================================================
    std::vector<float> hazard_rate(in.num_counterparties);
    std::vector<float> fx_net_nominal(in.num_counterparties, 0.0f);
    std::vector<float> usd_fixed(in.num_counterparties * in.tenor[1]*12, 0.0f);
    std::vector<float> eur_fixed(in.num_counterparties * in.tenor[1]*12, 0.0f);
    std::vector<float> usd_floating(in.num_counterparties * in.tenor[1]*12, 0.0f);
    std::vector<float> eur_floating(in.num_counterparties * in.tenor[1]*12, 0.0f);

    // Counterparties, deals and netting sets simulation
    LOG(INFO) << "Simulating Bank Data=================";
    // Generate counterparties
    LOG(INFO) << "Generating " << in.num_counterparties << " Counterparties...";
    std::vector<unsigned> idx[5]; // these vectors help us to find where is the ith A rating counterparty
    std::vector<Counterparty> counterparties = create_counterparties(in, hazard_rate, idx);
    // Generate forex deals
    LOG(INFO) << "Generating " << in.num_fx << " Forex EUR/USD Deals...";
    std::vector<Swap> fx_deals = create_fxs(in);
    // Generate swap deals
    LOG(INFO) << "Generating " << in.num_swap << " Swap Deals...";
    std::vector<Swap> swap_deals = create_swaps(in);
    // Netting sets
    LOG(INFO) << "Allocating Deals to Counterparties...";
    allocate(counterparties, fx_deals, swap_deals, in, idx, fx_net_nominal, usd_fixed, eur_fixed, usd_floating, eur_floating);

    std::ofstream fout("output.data");
    boost::archive::binary_oarchive oar(fout);
    oar << hazard_rate << fx_net_nominal << usd_fixed << eur_fixed << usd_floating << eur_floating;

    if (LogFile::ReportLevel() >= DIAG) {
        LOG(DIAG) << "Counterparty 0 EUR Swap : Net Fixed-leg Cash Flow at each time step (month)"; // Only log the first counterparty EUR Swap
        for (unsigned i = 0; i < in.tenor[1]*12; ++i)
        { LOG(DIAG) << "     " << i + 1 << " : " << eur_fixed[i]; }
        for (unsigned i = 0; i < in.num_counterparties; ++i)
        { LOG(DIAG) << "Forex Net Nominal of Counterparty " << i << " : " << fx_net_nominal[i]; }
    }


//===================================================================================
// Combine OpenMP and CUDA Thrust to do multi-GPU Monte Carlo
//===================================================================================
  Timer timer; // Set up timer
  timer.start();
  LOG(INFO) << "Monte Carlo Parameters===============";
  LOG(INFO) << "Number of Paths : " << in.num_paths;
  LOG(INFO) << "Number of Years : " << in.tenor[1];
  LOG(INFO) << "Number of Steps per Year : " << 12.;
  LOG(INFO) << "Number of Steps along a Path : " << in.tenor[1]*12;
  float dt = 1.0f / 12.;
  float mean_fx = (in.fx_drift - in.fx_sigma * in.fx_sigma / 2.0f) * dt; // (r - sigma^2 / 2) * dt
  float stdev_fx = in.fx_sigma * sqrt(dt); // sigma * sqrt(dt)

#ifdef _GPU_ // Monte Carlo on multi-GPU
  LOG(INFO) << "Monte Carlo on Multi-GPU=============";
  thrust::host_vector<float> hCVA[num_gpus]; // final CVA numbers for each counterparty end up in hCVA[0]
  for (unsigned j = 0; j < num_gpus; ++j) { hCVA[j].resize(in.num_counterparties); }
  omp_set_num_threads(num_gpus);
  #pragma omp parallel
  {
    using namespace thrust;
    Timer loctimer; // Set up local timer
    loctimer.start();
    int i = omp_get_thread_num();
    hipSetDevice(i);

    // Copy data from host
    int size0 = in.tenor[1]*12, size1 = in.num_counterparties, size2 = in.tenor[1]*12 * in.num_counterparties;
    device_vector<float> dFX_Price(size0); // Future price of Forex EUR/USD from time 0 to in.tenor[1] - dt
    device_vector<float> dUSD_Curve[4], dEUR_Curve[4]; // Curve parameters from time 0 to in.tenor[1] - dt
    for (unsigned j = 0; j < 4; ++j) { dUSD_Curve[j].resize(size0); dEUR_Curve[j].resize(size0); }
    device_vector<float> dHazard_rate(size1), dFX_nominal(size1), dSwap(size1), dCVA(size1, 0.0f);
    device_vector<float> dUSD_fixed(size2), dEUR_fixed(size2), dUSD_floating(size2), dEUR_floating(size2), dIdx(size2);
    counting_iterator<unsigned> beg(0), end0(size0), end2(size2);
    thrust::transform(beg, end2, dIdx.begin(), getIdx(in.tenor[1]*12));
    thrust::copy(fx_net_nominal.begin(), fx_net_nominal.end(), dFX_nominal.begin());
    thrust::copy(hazard_rate.begin(), hazard_rate.end(), dHazard_rate.begin());
    thrust::copy(usd_fixed.begin(), usd_fixed.end(), dUSD_fixed.begin());
    thrust::copy(eur_fixed.begin(), eur_fixed.end(), dEUR_fixed.begin());
    thrust::copy(usd_floating.begin(), usd_floating.end(), dUSD_floating.begin());
    thrust::copy(eur_floating.begin(), eur_floating.end(), dEUR_floating.begin());
    typedef thrust::device_vector<float>::iterator FloatIterator;
    typedef thrust::tuple<FloatIterator, FloatIterator, FloatIterator, FloatIterator> TupleFloatIterator;
    typedef thrust::zip_iterator<TupleFloatIterator> ZipIterator;
    ZipIterator first = make_zip_iterator(make_tuple(dSwap.begin(), dFX_nominal.begin(), dHazard_rate.begin(), dCVA.begin()));
    ZipIterator last = make_zip_iterator(make_tuple(dSwap.end(), dFX_nominal.end(), dHazard_rate.end(), dCVA.end()));

    // Figure out the number of paths this gpu is going to deal with
    int num_paths = (i == 0) ? in.num_paths - in.num_paths / num_gpus * (num_gpus - 1) : in.num_paths / num_gpus;
    LOG(INFO) << "   GPU " << i << " : " << num_paths << " paths";

    for (unsigned p = 0; p < num_paths; ++p) {
      // Simulate a path
      // We need 1000000i to avoid the same seed in different GPUs
      // We need 10p to make sure seeds are different in different paths (9 rng constructors per path)
      int seed = 1000000 * i + 10 * p;
      transform_exclusive_scan(beg, end0, dFX_Price.begin(), random_lognormal(mean_fx, stdev_fx, seed), in.fx_init, multiplies<float>());
      // Curve parameters may go negative since we use OU process but not CIR model
      for (unsigned j = 0; j < 4; ++j) {
        transform_exclusive_scan(beg, end0, dUSD_Curve[j].begin(), getA(in.usd_params[j][2], dt, seed+2*j+1), 0, plus<float>());
        transform_exclusive_scan(beg, end0, dEUR_Curve[j].begin(), getA(in.eur_params[j][2], dt, seed+2*j+2), 0, plus<float>());
        thrust::transform(make_zip_iterator(make_tuple(dUSD_Curve[j].begin(), beg)),
            make_zip_iterator(make_tuple(dUSD_Curve[j].end(), end0)), dUSD_Curve[j].begin(), getCurve(in.usd_params[j], dt));
        thrust::transform(make_zip_iterator(make_tuple(dEUR_Curve[j].begin(), beg)),
            make_zip_iterator(make_tuple(dEUR_Curve[j].end(), end0)), dEUR_Curve[j].begin(), getCurve(in.eur_params[j], dt));
      }
      if (LogFile::ReportLevel() >= DIAG) {
          if ((i == 0) && (p == 0)) {
              for (int j = 0; j < in.tenor[1]*12; ++j)
              { LOG(DIAG) << "   GPU " << i << " : USD Curve Beta1 (Step " << j << ") = " << dUSD_Curve[1][j]; }
          }
      }
      
      // For each counterparty, accumulate CVA along the path
      for (unsigned j = 0; j < in.tenor[1]*12; ++j) {
        priceSwap pricer(dUSD_Curve[0][j], dUSD_Curve[1][j], dUSD_Curve[2][j], dUSD_Curve[3][j],
                         dEUR_Curve[0][j], dEUR_Curve[1][j], dEUR_Curve[2][j], dEUR_Curve[3][j], in.tenor[1]*12, j, dt, dFX_Price[j]);
        reduce_by_key(dIdx.begin(), dIdx.end(), make_transform_iterator(make_zip_iterator(make_tuple(beg, dUSD_fixed.begin(), dEUR_fixed.begin(),
                      dUSD_floating.begin(), dUSD_floating.begin())), pricer), make_discard_iterator(), dSwap.begin());
        thrust::transform(first, last, dCVA.begin(), accumulateCVA(j, dFX_Price[j], in.disc_rate, dt));
      }
    } // path loop

    thrust::copy(dCVA.begin(), dCVA.end(), hCVA[i].begin());
    loctimer.end();
    LOG(INFO) << "Time (GPU " << i << ") : " << std::setprecision(3) << std::fixed << loctimer.d_time << " s";
  } // omp parallel

  for (unsigned j = 1; j < num_gpus; ++j) {
      thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(hCVA[0].begin(), hCVA[j].begin())),
          thrust::make_zip_iterator(thrust::make_tuple(hCVA[0].end(), hCVA[j].end())), hCVA[0].begin(), add());
  }
  thrust::for_each(hCVA[0].begin(), hCVA[0].end(), divide(in.num_paths));
  float aggregateCVA = thrust::reduce(hCVA[0].begin(), hCVA[0].end());
  LOG(OFF) << "Aggregate CVA for the bank : " << aggregateCVA;
  if (LogFile::ReportLevel() >= DIAG) {
      for (int j = 0; j < in.num_counterparties; ++j)
      { LOG(DIAG) << "CVA for Counterparty " << j << " (hazard rate " << hazard_rate[j] << ") : " << hCVA[0][j]; }
  }

#endif

#ifdef _CPU_
    // Prepare data
    LOG(INFO) << "Preparing Data=======================";
    thrust::host_vector<float> CVA(in.num_counterparties, 0.0f);  // the final CVA numbers end up in this vector
    typedef thrust::device_vector<float>::iterator FloatIterator;
    typedef thrust::tuple<FloatIterator, FloatIterator, FloatIterator> TupleFloatIterator3;
    typedef thrust::zip_iterator<TupleFloatIterator3> ZipIterator;
    LOG(INFO) << "Initializing Data for CPUs===========";
    thrust::device_vector<float> dFX_net_nominal = fx_net_nominal;
    thrust::device_vector<float> dFX_price(in.tenor[1]*12);
    thrust::device_vector<float> dCVA(in.num_counterparties, 0.0f);
    thrust::device_vector<float> dHazard_rate = hazard_rate;
    // Put the net nominal, hazard rate and CVA together
    ZipIterator first = thrust::make_zip_iterator(thrust::make_tuple(dFX_net_nominal.begin(), dHazard_rate.begin(), dCVA.begin()));
    ZipIterator last  = thrust::make_zip_iterator(thrust::make_tuple(dFX_net_nominal.end(), dHazard_rate.end(), dCVA.begin()));
    for (unsigned p = 0; p < in.num_paths; ++p) {
        // Simulate a path
        // It's possible that two iterations are in the same second. That's why we need +p in the constructor of random_normal.
        thrust::transform_inclusive_scan(thrust::counting_iterator<unsigned>(0), thrust::counting_iterator<unsigned>(in.tenor[1]*12),
                                         dFX_price.begin(), random_normal(mean_fx, stdev_fx, p), thrust::plus<float>());
        thrust::transform(dFX_price.begin(), dFX_price.end(), dFX_price.begin(), generatePrice(in.fx_init));
        // For each counterparty, accumulate CVA along the path
        for (unsigned j = 0; j < in.tenor[1]*12; ++j) {
            thrust::transform(first, last, dCVA.begin(), incrementCVA(in.disc_rate, dFX_price[j], (j + 1) * dt, dt));
        }
        LOG(DIAG) << "Path " << p + 1 << " Finishs...";
    }
    thrust::copy(thrust::make_transform_iterator(dCVA.begin(), normalizeCVA(in.num_paths)),
                 thrust::make_transform_iterator(dCVA.end(), normalizeCVA(in.num_paths)), CVA.begin());
    // Calculate aggregate CVA
    float aggregateCVA = thrust::reduce(CVA.begin(), CVA.end());
    LOG(OFF) << "Aggregate CVA for the bank : " << aggregateCVA;
#endif

    timer.end();
    LOG(OFF) << "Time for Path Simulation and CVA Calculation : " << std::setprecision(3) << std::fixed << timer.d_time << " s";
    LOG(OFF) << argv[0] << " Ends Here...====================================";
    return 0;
}
